//========================================================================================================================================================================================================200
//	DEFINE/INCLUDE
//========================================================================================================================================================================================================200

//======================================================================================================================================================150
//	MAIN FUNCTION HEADER
//======================================================================================================================================================150

#include "./../lavaMD.h"								// (in the main program folder)	needed to recognized input parameters

//======================================================================================================================================================150
//	UTILITIES
//======================================================================================================================================================150

#include "./../util/timer/timer.h"					// (in library path specified to compiler)	needed by timer
#include "cudacommon.h"

//======================================================================================================================================================150
//	KERNEL_GPU_CUDA_WRAPPER FUNCTION HEADER
//======================================================================================================================================================150

#include "./kernel_gpu_cuda_wrapper.h"				// (in the current directory)

//======================================================================================================================================================150
//	KERNEL
//======================================================================================================================================================150

#include "./kernel_gpu_cuda.cu"						// (in the current directory)	GPU kernel, cannot include with header file because of complications with passing of constant memory variables

//========================================================================================================================================================================================================200
//	KERNEL_GPU_CUDA_WRAPPER FUNCTION
//========================================================================================================================================================================================================200

#define MAX_STREAM 32

/// <summary>	An enum constant representing the void option. </summary>
void 
kernel_gpu_cuda_wrapper(par_str par_cpu,
						dim_str dim_cpu,
						box_str* box_cpu,
						FOUR_VECTOR* rv_cpu,
						fp* qv_cpu,
						FOUR_VECTOR* fv_cpu,
                        ResultDatabase &resultDB,
						OptionParser &op,
                        ofstream &ofile,
                        sem_t *sem)
{
	bool uvm = op.getOptionBool("uvm");
	bool zero_copy = op.getOptionBool("zero-copy");
	bool uvm_prefetch = op.getOptionBool("uvm-prefetch");
	bool copy = op.getOptionBool("copy");
	bool pageable = op.getOptionBool("pageable");
	bool async = op.getOptionBool("async");
	bool pud = op.getOptionBool("pud");
    const bool is_barrier = op.getOptionBool("sem");
    string bench_name = op.getOptionString("bench");

    float kernelTime = 0.0f;
    float transferTime = 0.0f;
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    float elapsedTime;
    int device = 0;
    checkCudaErrors(hipGetDevice(&device));

    int s_id = 0;
    hipStream_t streams[MAX_STREAM];
    if (async) {
        for (int s = 0; s < MAX_STREAM; s++) {
            hipStreamCreate(&streams[s]);
        }
    }

	//======================================================================================================================================================150
	//	CPU VARIABLES
	//======================================================================================================================================================150

	//======================================================================================================================================================150
	//	GPU SETUP
	//======================================================================================================================================================150

	//====================================================================================================100
	//	INITIAL DRIVER OVERHEAD
	//====================================================================================================100

	checkCudaErrors(hipDeviceSynchronize());

	//====================================================================================================100
	//	VARIABLES
	//====================================================================================================100

	box_str* d_box_gpu;
	FOUR_VECTOR* d_rv_gpu;
	fp* d_qv_gpu;
	FOUR_VECTOR* d_fv_gpu;

	dim3 threads;
	dim3 blocks;

	//====================================================================================================100
	//	EXECUTION PARAMETERS
	//====================================================================================================100

	blocks.x = dim_cpu.number_boxes;
	blocks.y = 1;
	threads.x = NUMBER_THREADS;											// define the number of threads in the block
	threads.y = 1;

	//======================================================================================================================================================150
	//	GPU MEMORY				(MALLOC)
	//======================================================================================================================================================150

	//====================================================================================================100
	//	GPU MEMORY				(MALLOC) COPY IN
	//====================================================================================================100

	//==================================================50
	//	boxes
	//==================================================50

	if (uvm || uvm_prefetch || zero_copy || pud) {
		d_box_gpu = box_cpu;
	} else if (copy) {
		checkCudaErrors(hipMalloc(	(void **)&d_box_gpu,
					dim_cpu.box_mem));
	} else {
		checkCudaErrors(hipMalloc(	(void **)&d_box_gpu,
					dim_cpu.box_mem));
    }

	//==================================================50
	//	rv
	//==================================================50

	if (uvm || uvm_prefetch || zero_copy || pud) {
		d_rv_gpu = rv_cpu;
	} else if (copy) {
		checkCudaErrors(hipMalloc(	(void **)&d_rv_gpu, 
					dim_cpu.space_mem));
	} else {
		checkCudaErrors(hipMalloc(	(void **)&d_rv_gpu, 
					dim_cpu.space_mem));
    }

	//==================================================50
	//	qv
	//==================================================50

	if (uvm || uvm_prefetch || zero_copy || pud) {
		d_qv_gpu = qv_cpu;
	} else if (copy) {
		checkCudaErrors(hipMalloc(	(void **)&d_qv_gpu,
					dim_cpu.space_mem2));
	} else {
		checkCudaErrors(hipMalloc(	(void **)&d_qv_gpu,
					dim_cpu.space_mem2));
    }

	//====================================================================================================100
	//	GPU MEMORY				(MALLOC) COPY
	//====================================================================================================100

	//==================================================50
	//	fv
	//==================================================50

	if (uvm || uvm_prefetch || zero_copy || pud) {
		d_fv_gpu = fv_cpu;
	} else if (copy) {
		checkCudaErrors(hipMalloc(	(void **)&d_fv_gpu, 
					dim_cpu.space_mem));
	} else {
		checkCudaErrors(hipMalloc(	(void **)&d_fv_gpu, 
					dim_cpu.space_mem));
    }

	//======================================================================================================================================================150
	//	GPU MEMORY			COPY
	//======================================================================================================================================================150

	//====================================================================================================100
	//	GPU MEMORY				(MALLOC) COPY IN
	//====================================================================================================100

	//==================================================50
	//	boxes
	//==================================================50

    if (!pageable && !copy)
        checkCudaErrors(hipEventRecord(start, 0));

	if (uvm) {
		// Demand paging
	} else if (pud) {
        //checkCudaErrors(hipMemAdvise(d_box_gpu, dim_cpu.box_mem, hipMemAdviseSetReadMostly, 0));
    } else if (zero_copy) {
        checkCudaErrors(hipMemAdvise(d_box_gpu, dim_cpu.box_mem, hipMemAdviseSetAccessedBy, 0));
    } else if (uvm_prefetch) {
        checkCudaErrors(hipMemPrefetchAsync(d_box_gpu, dim_cpu.box_mem, device));
    } else if (copy || pageable) {
        if (is_barrier && pageable) {
            int sval;
            sem_post(sem);
            sem_getvalue(sem, &sval);
            while (sval == 1) {
                sem_getvalue(sem, &sval);
            }
            printf("[Barrier] Copying starts\n");
        }
        checkCudaErrors(hipEventRecord(start, 0));
        if (async) {
            checkCudaErrors(hipMemcpyAsync(d_box_gpu, 
                        box_cpu,
                        dim_cpu.box_mem, 
                        hipMemcpyHostToDevice, streams[s_id++]));
        } else {
            checkCudaErrors(hipMemcpy(	d_box_gpu, 
                        box_cpu,
                        dim_cpu.box_mem, 
                        hipMemcpyHostToDevice));
        }
	} 

	//==================================================50
	//	rv
	//==================================================50
	
	if (uvm) {
		// Demand paging
	} else if (zero_copy || pud) {
        //checkCudaErrors(hipMemAdvise(d_rv_gpu, dim_cpu.space_mem, hipMemAdviseSetAccessedBy, 0));
    } else if (uvm_prefetch) {
        checkCudaErrors(hipMemPrefetchAsync(d_rv_gpu, dim_cpu.space_mem, device));
    } else if (copy || pageable) {
        if (async) {
            checkCudaErrors(hipMemcpyAsync(d_rv_gpu,
                        rv_cpu,
                        dim_cpu.space_mem,
                        hipMemcpyHostToDevice, streams[s_id++]));
        } else {
            checkCudaErrors(hipMemcpy(	d_rv_gpu,
                        rv_cpu,
                        dim_cpu.space_mem,
                        hipMemcpyHostToDevice));

        }
	} 

	//==================================================50
	//	qv
	//==================================================50

	if (uvm) {
		// Demand paging
	} else if (zero_copy || pud) {
        //checkCudaErrors(hipMemAdvise(d_qv_gpu, dim_cpu.space_mem2, hipMemAdviseSetAccessedBy, device));
    } else if (uvm_prefetch) {
        checkCudaErrors(hipMemPrefetchAsync(d_qv_gpu, dim_cpu.space_mem2, device));
    } else if (copy || pageable) {
        if (async) {
            checkCudaErrors(hipMemcpyAsync(d_qv_gpu,
                        qv_cpu,
                        dim_cpu.space_mem2,
                        hipMemcpyHostToDevice, streams[s_id++]));
        } else {
            checkCudaErrors(hipMemcpy(	d_qv_gpu,
                        qv_cpu,
                        dim_cpu.space_mem2,
                        hipMemcpyHostToDevice));

        }
	} 
	//====================================================================================================100
	//	GPU MEMORY				(MALLOC) COPY
	//====================================================================================================100

	//==================================================50
	//	fv
	//==================================================50

	if (uvm) {
		// Demand paging
	} else if (zero_copy) {
        checkCudaErrors(hipMemAdvise(d_fv_gpu, dim_cpu.space_mem, hipMemAdviseSetAccessedBy, 0));
    } else if (uvm_prefetch) {
        checkCudaErrors(hipMemPrefetchAsync(d_fv_gpu, dim_cpu.space_mem, device));
    } else if (copy || pageable) {
        if (async) {
            checkCudaErrors(hipMemcpyAsync(d_fv_gpu, 
                        fv_cpu, 
                        dim_cpu.space_mem, 
                        hipMemcpyHostToDevice, streams[s_id++]));
        } else {
            checkCudaErrors(hipMemcpy(	d_fv_gpu, 
                        fv_cpu, 
                        dim_cpu.space_mem, 
                        hipMemcpyHostToDevice));
        }
	} 
	checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
    transferTime += elapsedTime * 1.e-3;

	//======================================================================================================================================================150
	//	KERNEL
	//======================================================================================================================================================150

    // (taeklim): Waiting for the other apps finishes the initialization
    if (is_barrier && uvm) {
        int sval;
        sem_post(sem);
        sem_getvalue(sem, &sval);
        while (sval == 1) {
            sem_getvalue(sem, &sval);
        }
        printf("[Barrier] Kernel starts\n");
    }

	// launch kernel - all boxes
    checkCudaErrors(hipEventRecord(start, 0));
	kernel_gpu_cuda<<<blocks, threads>>>(	par_cpu,
											dim_cpu,
											d_box_gpu,
											d_rv_gpu,
											d_qv_gpu,
											d_fv_gpu);
	checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
    kernelTime += elapsedTime * 1.e-3;

    CHECK_CUDA_ERROR();
	checkCudaErrors(hipDeviceSynchronize());

	//======================================================================================================================================================150
	//	GPU MEMORY			COPY (CONTD.)kernel
	//======================================================================================================================================================150

    checkCudaErrors(hipEventRecord(start, 0));

	if (uvm || uvm_prefetch || zero_copy) {
		checkCudaErrors(hipMemPrefetchAsync(d_fv_gpu, dim_cpu.space_mem, hipCpuDeviceId));
        checkCudaErrors(hipStreamSynchronize(0));
	} else if (copy || pageable) {
        if (async) {
            checkCudaErrors(hipMemcpyAsync(fv_cpu, 
                        d_fv_gpu,
                        dim_cpu.space_mem, 
                        hipMemcpyDeviceToHost, streams[s_id++]));
        } else {
            checkCudaErrors(hipMemcpy(	fv_cpu, 
                        d_fv_gpu,
                        dim_cpu.space_mem, 
                        hipMemcpyDeviceToHost));
        }
	} 
	checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
    transferTime += elapsedTime * 1.e-3;

    char atts[1024];
    sprintf(atts, "boxes1d:%d", dim_cpu.boxes1d_arg);
    resultDB.AddResult("lavamd_kernel_time", atts, "sec", kernelTime);
    //resultDB.AddResult("lavamd_transfer_time", atts, "sec", transferTime);
    resultDB.AddResult("lavamd_total_time", atts, "sec", kernelTime + transferTime);
    //resultDB.AddResult("lavamd_parity", atts, "N", transferTime / kernelTime);
    ofile << bench_name << ", " << kernelTime + transferTime << ", " << endl;

    if (async) {
        for (int s = 0; s < MAX_STREAM; s++) {
            hipStreamDestroy(streams[s]);
        }
    }

	//======================================================================================================================================================150
	//	GPU MEMORY DEALLOCATION
	//======================================================================================================================================================150

	if (uvm) {
		// Demand paging, no need to free
	} else if (uvm_prefetch || zero_copy || pud) {

	} else if (copy) {
		checkCudaErrors(hipFree(d_rv_gpu));
		checkCudaErrors(hipFree(d_qv_gpu));
		checkCudaErrors(hipFree(d_fv_gpu));
		checkCudaErrors(hipFree(d_box_gpu));
	} else {
		checkCudaErrors(hipFree(d_rv_gpu));
		checkCudaErrors(hipFree(d_qv_gpu));
		checkCudaErrors(hipFree(d_fv_gpu));
		checkCudaErrors(hipFree(d_box_gpu));
    }
}
