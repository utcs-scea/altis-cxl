////////////////////////////////////////////////////////////////////////////////////////////////////
// file:	altis\src\cuda\level2\nw\needle.cu
//
// summary:	Needle class
// 
// origin: Rodinia (http://rodinia.cs.virginia.edu/doku.php)
////////////////////////////////////////////////////////////////////////////////////////////////////

#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
#include <limits.h>

#include "cudacommon.h"
#include "OptionParser.h"
#include "ResultDatabase.h"
#include "needle.h"
#include "needle_kernel.cu"

#define LIMIT -999
#define SEED 7

int max_rows, max_cols, penalty;

void runTest(ResultDatabase &resultDB, OptionParser &op, ofstream &ofile, sem_t *sem);

/// <summary>	The blosum 62[24][24]. </summary>
int blosum62[24][24] = {{4,  -1, -2, -2, 0, -1, -1, 0, -2, -1, -1, -1,
                         -1, -2, -1, 1,  0, -3, -2, 0, -2, -1, 0,  -4},
                        {-1, 5,  0,  -2, -3, 1,  0,  -2, 0,  -3, -2, 2,
                         -1, -3, -2, -1, -1, -3, -2, -3, -1, 0,  -1, -4},
                        {-2, 0,  6,  1, -3, 0,  0,  0,  1, -3, -3, 0,
                         -2, -3, -2, 1, 0,  -4, -2, -3, 3, 0,  -1, -4},
                        {-2, -2, 1,  6, -3, 0,  2,  -1, -1, -3, -4, -1,
                         -3, -3, -1, 0, -1, -4, -3, -3, 4,  1,  -1, -4},
                        {0,  -3, -3, -3, 9,  -3, -4, -3, -3, -1, -1, -3,
                         -1, -2, -3, -1, -1, -2, -2, -1, -3, -3, -2, -4},
                        {-1, 1,  0,  0, -3, 5,  2,  -2, 0, -3, -2, 1,
                         0,  -3, -1, 0, -1, -2, -1, -2, 0, 3,  -1, -4},
                        {-1, 0,  0,  2, -4, 2,  5,  -2, 0, -3, -3, 1,
                         -2, -3, -1, 0, -1, -3, -2, -2, 1, 4,  -1, -4},
                        {0,  -2, 0,  -1, -3, -2, -2, 6,  -2, -4, -4, -2,
                         -3, -3, -2, 0,  -2, -2, -3, -3, -1, -2, -1, -4},
                        {-2, 0,  1,  -1, -3, 0,  0, -2, 8, -3, -3, -1,
                         -2, -1, -2, -1, -2, -2, 2, -3, 0, 0,  -1, -4},
                        {-1, -3, -3, -3, -1, -3, -3, -4, -3, 4,  2,  -3,
                         1,  0,  -3, -2, -1, -3, -1, 3,  -3, -3, -1, -4},
                        {-1, -2, -3, -4, -1, -2, -3, -4, -3, 2,  4,  -2,
                         2,  0,  -3, -2, -1, -2, -1, 1,  -4, -3, -1, -4},
                        {-1, 2,  0,  -1, -3, 1,  1,  -2, -1, -3, -2, 5,
                         -1, -3, -1, 0,  -1, -3, -2, -2, 0,  1,  -1, -4},
                        {-1, -1, -2, -3, -1, 0,  -2, -3, -2, 1,  2,  -1,
                         5,  0,  -2, -1, -1, -1, -1, 1,  -3, -1, -1, -4},
                        {-2, -3, -3, -3, -2, -3, -3, -3, -1, 0,  0,  -3,
                         0,  6,  -4, -2, -2, 1,  3,  -1, -3, -3, -1, -4},
                        {-1, -2, -2, -1, -3, -1, -1, -2, -2, -3, -3, -1,
                         -2, -4, 7,  -1, -1, -4, -3, -2, -2, -1, -2, -4},
                        {1,  -1, 1,  0, -1, 0,  0,  0,  -1, -2, -2, 0,
                         -1, -2, -1, 4, 1,  -3, -2, -2, 0,  0,  0,  -4},
                        {0,  -1, 0,  -1, -1, -1, -1, -2, -2, -1, -1, -1,
                         -1, -2, -1, 1,  5,  -2, -2, 0,  -1, -1, 0,  -4},
                        {-3, -3, -4, -4, -2, -2, -3, -2, -2, -3, -2, -3,
                         -1, 1,  -4, -3, -2, 11, 2,  -3, -4, -3, -2, -4},
                        {-2, -2, -2, -3, -2, -1, -2, -3, 2,  -1, -1, -2,
                         -1, 3,  -3, -2, -2, 2,  7,  -1, -3, -2, -1, -4},
                        {0, -3, -3, -3, -1, -2, -2, -3, -3, 3,  1,  -2,
                         1, -1, -2, -2, 0,  -3, -1, 4,  -3, -2, -1, -4},
                        {-2, -1, 3,  4, -3, 0,  1,  -1, 0, -3, -4, 0,
                         -3, -3, -2, 0, -1, -4, -3, -3, 4, 1,  -1, -4},
                        {-1, 0,  0,  1, -3, 3,  4,  -2, 0, -3, -3, 1,
                         -1, -3, -1, 0, -1, -3, -2, -2, 1, 4,  -1, -4},
                        {0,  -1, -1, -1, -2, -1, -1, -1, -1, -1, -1, -1,
                         -1, -1, -2, 0,  0,  -2, -1, -1, -1, -1, -1, -4},
                        {-4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4,
                         -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, 1}};


////////////////////////////////////////////////////////////////////////////////////////////////////
/// <summary>	Add benchmark specific options parsing.  The user is allowed to specify
/// the size of the input data in kiB.. </summary>
///
/// <remarks>	Ed, 5/20/2020.
/// 			Anthony Danalis, 9/08, 2009
///
/// <param name="op">	[in,out] the options parser / parameter database. </param>
////////////////////////////////////////////////////////////////////////////////////////////////////

void addBenchmarkSpecOptions(OptionParser &op) {
  op.addOption("uvm", OPT_BOOL, "0", "enable CUDA Unified Virtual Memory, only demand paging");
  op.addOption("dimensions", OPT_INT, "0", "dimensions");
  op.addOption("penalty", OPT_INT, "10", "penalty");
  op.addOption("resultsfile", OPT_STRING, "", "file to write results to");
}


////////////////////////////////////////////////////////////////////////////////////////////////////
/// <summary>	Executes the benchmark operation. </summary>
///
/// <remarks>	Ed, 5/20/2020.
/// 			Kyle Spafford, 8/13/2009 </remarks>
///
/// <param name="resultDB">	[in,out] results from the benchmark are stored in this db. </param>
/// <param name="op">	   	[in,out] the options parser / parameter database. </param>
////////////////////////////////////////////////////////////////////////////////////////////////////

//void RunBenchmark(ResultDatabase &resultDB, OptionParser &op) {
void RunBenchmark(ResultDatabase &resultDB, OptionParser &op, ofstream &ofile, sem_t *sem) {
  printf("Running Needleman-Wunsch\n");

  int device;
  hipGetDevice(&device);
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, device);

  bool quiet = op.getOptionBool("quiet");
  int dim = op.getOptionInt("dimensions");
  penalty = op.getOptionInt("penalty");

  if(penalty < 0) {
      printf("Error: Penalty must be a positive number.\n");
      return;
  }
  if(dim < 0) {
      printf("Error: Dimensions must be positive.\n");
      return;
  }

  if (dim == 0) {
    int problemSizes[4] = {100, 1000, 6000, 40000};
    dim = problemSizes[op.getOptionInt("size") - 1];
  }

  long long num_items = (long long)dim * (long long)dim;
  if(num_items >= INT_MAX) {
      printf("Error: Total size cannot exceed INT_MAX");
      return;
  }

  if(!quiet) {
      printf("WG size of kernel = %d \n", BLOCK_SIZE);
      printf("Max Rows x Cols: %dx%d\n", dim, dim);
      printf("Penalty: %d\n\n", penalty);
  }
  srand(SEED);

  int passes = op.getOptionInt("passes");
  for (int i = 0; i < passes; i++) {
      if(!quiet) {
          printf("Pass %d: ", i);
      }
      max_rows = dim;
      max_cols = dim;
      runTest(resultDB, op, ofile, sem);
      if(!quiet) {
          printf("Done.\n");
      }
  }
}

////////////////////////////////////////////////////////////////////////////////////////////////////
/// <summary>	Executes the test operation. </summary>
///
/// <remarks>	Ed, 5/20/2020. </remarks>
///
/// <param name="resultDB">	[in,out] The result database. </param>
/// <param name="op">	   	[in,out] The operation. </param>
////////////////////////////////////////////////////////////////////////////////////////////////////

void runTest(ResultDatabase &resultDB, OptionParser &op, ofstream &ofile, sem_t *sem) {
  bool uvm = op.getOptionBool("uvm");
  bool uvm_prefetch = op.getOptionBool("uvm-prefetch");
  bool copy = op.getOptionBool("copy");
  bool quiet = op.getOptionBool("quiet");
  bool pageable = op.getOptionBool("pageable");
  int *input_itemsets, *output_itemsets, *referrence;
  int *matrix_cuda, *referrence_cuda;
  int size;
  int device = 0;
  checkCudaErrors(hipGetDevice(&device));
  const bool is_barrier = op.getOptionBool("sem");
  string bench_name = op.getOptionString("bench");

  max_rows = max_rows + 1;
  max_cols = max_cols + 1;
  
  if (uvm || uvm_prefetch) {
    checkCudaErrors(hipMallocManaged(&referrence, max_rows * max_cols * sizeof(int)));
    checkCudaErrors(hipMallocManaged(&input_itemsets, max_rows * max_cols * sizeof(int)));
  } else if (copy) {
    checkCudaErrors(hipHostMalloc(&referrence, max_rows * max_cols * sizeof(int)));
    //checkCudaErrors(hipHostAlloc(&referrence, max_rows * max_cols * sizeof(int), hipHostMallocDefault));
    assert(referrence);
    checkCudaErrors(hipHostMalloc(&input_itemsets, max_rows * max_cols * sizeof(int)));
    assert(input_itemsets);
    checkCudaErrors(hipHostMalloc(&output_itemsets, max_rows * max_cols * sizeof(int)));
    assert(output_itemsets);
  } else if (pageable) {
    referrence = (int *)malloc(max_rows * max_cols * sizeof(int));
    assert(referrence);
    input_itemsets = (int *)malloc(max_rows * max_cols * sizeof(int));
    assert(input_itemsets);
    output_itemsets = (int *)malloc(max_rows * max_cols * sizeof(int));
    assert(output_itemsets);
  }

  if (!input_itemsets) {
      fprintf(stderr, "Error: Can not allocate memory\n");
      exit(0);
  }

  for (int i = 0; i < max_cols; i++) {
    for (int j = 0; j < max_rows; j++) {
      input_itemsets[i * max_cols + j] = 0;
    }
  }

  for (int i = 1; i < max_rows; i++) {  // please define your own sequence.
    input_itemsets[i * max_cols] = rand() % 10 + 1;
  }
  for (int j = 1; j < max_cols; j++) {  // please define your own sequence.
    input_itemsets[j] = rand() % 10 + 1;
  }

  for (int i = 1; i < max_cols; i++) {
    for (int j = 1; j < max_rows; j++) {
      referrence[i * max_cols + j] =
          blosum62[input_itemsets[i * max_cols]][input_itemsets[j]];
    }
  }

  for (int i = 1; i < max_rows; i++)
    input_itemsets[i * max_cols] = -i * penalty;
  for (int j = 1; j < max_cols; j++) input_itemsets[j] = -j * penalty;

  size = max_cols * max_rows;

  if (uvm || uvm_prefetch) {
    // Do nothing
  } else if (copy || pageable) {
    checkCudaErrors(hipMalloc((void **)&referrence_cuda, sizeof(int) * size));
    checkCudaErrors(hipMalloc((void **)&matrix_cuda, sizeof(int) * size));
  } 

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float elapsedTime;
  double transferTime = 0.;
  double kernelTime = 0;
  double totalTime = 0;

  // Notice that here we used demand paging so no cpy time included, could also use HyperQ
  if (uvm) {
      hipEventRecord(start, 0);
    referrence_cuda = referrence;
    matrix_cuda = input_itemsets;
  } else if (uvm_prefetch) {
      hipEventRecord(start, 0);
    referrence_cuda = referrence;
    matrix_cuda = input_itemsets;
      checkCudaErrors(hipMemPrefetchAsync(referrence_cuda, sizeof(int) * size , device));
      hipStream_t s1;
      checkCudaErrors(hipStreamCreate(&s1));
      checkCudaErrors(hipMemPrefetchAsync(matrix_cuda, sizeof(int) * size , device, s1));
      checkCudaErrors(hipStreamDestroy(s1));
  } else if (copy || pageable) {
    if (is_barrier && pageable) {
        int sval;
        sem_post(sem);
        sem_getvalue(sem, &sval);
        while (sval == 1) {
            sem_getvalue(sem, &sval);
        }
        printf("[Barrier] Copying starts\n");
    }
    hipEventRecord(start, 0);
    checkCudaErrors(hipMemcpy(referrence_cuda, referrence, sizeof(int) * size,
            hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(matrix_cuda, input_itemsets, sizeof(int) * size,
            hipMemcpyHostToDevice));
  } 
  printf("Done copying...\n");
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
  transferTime += elapsedTime * 1.e-3; // convert to seconds

  dim3 dimGrid;
  dim3 dimBlock(BLOCK_SIZE, 1);
  int block_width = (max_cols - 1) / BLOCK_SIZE;

  // (taeklim): Waiting for the other apps finishes the initialization
  if (is_barrier && uvm) {
      int sval;
      sem_post(sem);
      sem_getvalue(sem, &sval);
      while (sval == 1) {
          sem_getvalue(sem, &sval);
      }
      printf("[Barrier] Kernel starts\n");
  }
  hipEventRecord(start, 0);
  // process top-left matrix
  for (int i = 1; i <= block_width; i++) {
    dimGrid.x = i;
    dimGrid.y = 1;
    //hipEventRecord(start, 0);
    needle_cuda_shared_1<<<dimGrid, dimBlock>>>(
            referrence_cuda, matrix_cuda, max_cols, penalty, i, block_width);
    //hipEventRecord(stop, 0);
//    hipEventSynchronize(stop);
//    hipEventElapsedTime(&elapsedTime, start, stop);
//    kernelTime += elapsedTime * 1.e-3;
    CHECK_CUDA_ERROR();
  }
  // process bottom-right matrix
  for (int i = block_width - 1; i >= 1; i--) {
    dimGrid.x = i;
    dimGrid.y = 1;
//    hipEventRecord(start, 0);
    needle_cuda_shared_2<<<dimGrid, dimBlock>>>(
        referrence_cuda, matrix_cuda, max_cols, penalty, i, block_width);
//    hipEventRecord(stop, 0);
//    hipEventSynchronize(stop);
//    hipEventElapsedTime(&elapsedTime, start, stop);
//    kernelTime += elapsedTime * 1.e-3;
    CHECK_CUDA_ERROR();
  }

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
  kernelTime += elapsedTime * 1.e-3;

  printf("Done kernel...\n");

  hipEventRecord(start, 0);
  if (uvm || uvm_prefetch) {
    output_itemsets = matrix_cuda;
    checkCudaErrors(hipMemPrefetchAsync(output_itemsets, sizeof(int) * size, hipCpuDeviceId));
    checkCudaErrors(hipStreamSynchronize(0));
  } else if (copy || pageable) {
    checkCudaErrors(hipMemcpy(output_itemsets, matrix_cuda, sizeof(int) * size,
            hipMemcpyDeviceToHost));
  }
  printf("Done copying...\n");

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
  transferTime += elapsedTime * 1.e-3; // convert to seconds
  //kernelTime += elapsedTime * 1.e-3;
  totalTime += kernelTime + transferTime;

//  string outfile = op.getOptionString("outputFile");
//  if (outfile != "") {
////      FILE *fpo = fopen(outfile.c_str(), "w");
////      if(!quiet) {
////        fprintf(fpo, "Print traceback value GPU to %s:\n", outfile.c_str());
////      }
//
//      for (int i = max_rows - 2, j = max_rows - 2; i >= 0, j >= 0;) {
//          int nw, n, w, traceback;
//          if (i == max_rows - 2 && j == max_rows - 2) {
//              // print the first element
//              fprintf(fpo, "%d ", output_itemsets[i*max_cols+j]);
//          }
//          if (i == 0 && j == 0) {
//              break;
//          }
//          if (i > 0 && j > 0) {
//              nw = output_itemsets[(i - 1) * max_cols + j - 1];
//              w = output_itemsets[i * max_cols + j - 1];
//              n = output_itemsets[(i - 1) * max_cols + j];
//          } else if (i == 0) {
//              nw = n = LIMIT;
//              w = output_itemsets[i * max_cols + j - 1];
//          } else if (j == 0) {
//              nw = w = LIMIT;
//              n = output_itemsets[(i - 1) * max_cols + j];
//          } else {
//          }
//
//          // traceback = maximum(nw, w, n);
//          int new_nw, new_w, new_n;
//          new_nw = nw + referrence[i * max_cols + j];
//          new_w = w - penalty;
//          new_n = n - penalty;
//
//          traceback = maximum(new_nw, new_w, new_n);
//          if (traceback == new_nw) {
//              traceback = nw;
//          }
//          if (traceback == new_w) {
//              traceback = w;
//          }
//          if (traceback == new_n) {
//              traceback = n;
//          }
//
//          fprintf(fpo, "%d ", traceback);
//          if (traceback == nw) {
//              i--;
//              j--;
//              continue;
//          } else if (traceback == w) {
//              j--;
//              continue;
//          } else if (traceback == n) {
//              i--;
//              continue;
//          } else {
//          }
//      }
//      fclose(fpo);
//  }
  printf("Done output...\n");

  // Cleanup memory
  if (uvm || uvm_prefetch) {
    checkCudaErrors(hipFree(referrence_cuda));
    checkCudaErrors(hipFree(matrix_cuda));
  } else if (copy) {
    checkCudaErrors(hipFree(referrence_cuda));
    checkCudaErrors(hipFree(matrix_cuda));
    checkCudaErrors(hipHostFree(output_itemsets));
    checkCudaErrors(hipHostFree(referrence));
    checkCudaErrors(hipHostFree(input_itemsets));
  } else {
    checkCudaErrors(hipFree(referrence_cuda));
    checkCudaErrors(hipFree(matrix_cuda));
    free(referrence);
    free(input_itemsets);
    free(output_itemsets);
  }

  char tmp[32];
  sprintf(tmp, "%ditems", size);
  string atts = string(tmp);
  //resultDB.AddResult("NW-TransferTime", atts, "sec", transferTime);
  resultDB.AddResult("NW-KernelTime", atts, "sec", kernelTime);
  resultDB.AddResult("NW-TotalTime", atts, "sec", totalTime);
  ofile << bench_name << ", " << totalTime << ", " << endl;
//  resultDB.AddResult("NW-TotalTime", atts, "sec", transferTime + kernelTime);
//  resultDB.AddResult("NW-Rate_Parity", atts, "N", transferTime / kernelTime);
//  resultDB.AddOverall("Time", "sec", kernelTime+transferTime);
}
