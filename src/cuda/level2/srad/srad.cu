#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////////////////////////
// file:	altis\src\cuda\level2\srad\srad.cu
//
// summary:	Srad class
// 
// origin: Rodinia Benchmark (http://rodinia.cs.virginia.edu/doku.php)
////////////////////////////////////////////////////////////////////////////////////////////////////

#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "srad.h"

#include "OptionParser.h"
#include "ResultDatabase.h"
#include "cudacommon.h"

// includes, project
#include <hip/hip_runtime.h>

// includes, kernels
#include "srad_kernel.cu"

////////////////////////////////////////////////////////////////////////////////////////////////////
/// <summary>	A macro that defines seed. </summary>
///
/// <remarks>	Ed, 5/20/2020. </remarks>
////////////////////////////////////////////////////////////////////////////////////////////////////

#define SEED 7

/// <summary>	The kernel time. </summary>
float kernelTime = 0.0f;
/// <summary>	The transfer time. </summary>
float transferTime = 0.0f;

////////////////////////////////////////////////////////////////////////////////////////////////////
/// <summary>	Gets the stop. </summary>
///
/// <value>	The stop. </value>
////////////////////////////////////////////////////////////////////////////////////////////////////

hipEvent_t start, stop;
/// <summary>	The elapsed. </summary>
float elapsed;
/// <summary>	The check. </summary>
float *check;

////////////////////////////////////////////////////////////////////////////////////////////////////
/// <summary>	Random matrix. </summary>
///
/// <remarks>	Ed, 5/20/2020. </remarks>
///
/// <param name="I">   	[in,out] If non-null, zero-based index of the. </param>
/// <param name="rows">	The rows. </param>
/// <param name="cols">	The cols. </param>
////////////////////////////////////////////////////////////////////////////////////////////////////

void random_matrix(float *I, int rows, int cols);

////////////////////////////////////////////////////////////////////////////////////////////////////
/// <summary>	Executes the test operation. </summary>
///
/// <remarks>	Ed, 5/20/2020. </remarks>
///
/// <param name="argc">	The argc. </param>
/// <param name="argv">	[in,out] If non-null, the argv. </param>
////////////////////////////////////////////////////////////////////////////////////////////////////

void runTest(int argc, char **argv);

////////////////////////////////////////////////////////////////////////////////////////////////////
/// <summary>	Srads. </summary>
///
/// <remarks>	Ed, 5/20/2020. </remarks>
///
/// <param name="resultDB">   	[in,out] The result database. </param>
/// <param name="op">		  	[in,out] The operation. </param>
/// <param name="matrix">	  	[in,out] If non-null, the matrix. </param>
/// <param name="imageSize">  	Size of the image. </param>
/// <param name="speckleSize">	Size of the speckle. </param>
/// <param name="iters">	  	The iters. </param>
///
/// <returns>	A float. </returns>
////////////////////////////////////////////////////////////////////////////////////////////////////

float srad(ResultDatabase &resultDB, OptionParser &op, float* matrix, int imageSize, int speckleSize, int iters, ofstream &ofile, sem_t *sem);

////////////////////////////////////////////////////////////////////////////////////////////////////
/// <summary>	Srad gridsync. </summary>
///
/// <remarks>	Ed, 5/20/2020. </remarks>
///
/// <param name="resultDB">   	[in,out] The result database. </param>
/// <param name="op">		  	[in,out] The operation. </param>
/// <param name="matrix">	  	[in,out] If non-null, the matrix. </param>
/// <param name="imageSize">  	Size of the image. </param>
/// <param name="speckleSize">	Size of the speckle. </param>
/// <param name="iters">	  	The iters. </param>
///
/// <returns>	A float. </returns>
////////////////////////////////////////////////////////////////////////////////////////////////////

float srad_gridsync(ResultDatabase &resultDB, OptionParser &op, float* matrix, int imageSize, int speckleSize, int iters);

////////////////////////////////////////////////////////////////////////////////////////////////////
/// <summary>	Adds a benchmark specifier options. </summary>
///
/// <remarks>	Ed, 5/20/2020. </remarks>
///
/// <param name="op">	[in,out] The operation. </param>
////////////////////////////////////////////////////////////////////////////////////////////////////

void addBenchmarkSpecOptions(OptionParser &op) {
  op.addOption("imageSize", OPT_INT, "0", "image height and width");
  op.addOption("speckleSize", OPT_INT, "0", "speckle height and width");
  op.addOption("iterations", OPT_INT, "0", "iterations of algorithm");
}

////////////////////////////////////////////////////////////////////////////////////////////////////
/// <summary>	Executes the benchmark operation. </summary>
///
/// <remarks>	Ed, 5/20/2020. </remarks>
///
/// <param name="resultDB">	[in,out] The result database. </param>
/// <param name="op">	   	[in,out] The operation. </param>
////////////////////////////////////////////////////////////////////////////////////////////////////

//void RunBenchmark(ResultDatabase &resultDB, OptionParser &op) {
void RunBenchmark(ResultDatabase &resultDB, OptionParser &op, ofstream &ofile, sem_t *sem) {
  printf("Running SRAD\n");

  srand(SEED);
  bool quiet = op.getOptionBool("quiet");
  const bool uvm = op.getOptionBool("uvm");
  const bool zero_copy = op.getOptionBool("zero-copy");
  const bool pud = op.getOptionBool("pud");
  const bool copy = op.getOptionBool("copy");
  const bool pageable = op.getOptionBool("pageable");
  const bool uvm_advise = op.getOptionBool("uvm-advice");
  const bool uvm_prefetch = op.getOptionBool("uvm-prefetch");
  const bool uvm_prefetch_advise = op.getOptionBool("uvm-prefetch-advise");
  const bool coop = op.getOptionBool("coop");
  const bool is_barrier = op.getOptionBool("sem");
  int device = 0;
  checkCudaErrors(hipGetDevice(&device));

  // set parameters
  int imageSize = op.getOptionInt("imageSize");
  int speckleSize = op.getOptionInt("speckleSize");
  int iters = op.getOptionInt("iterations");
  if (imageSize == 0 || speckleSize == 0 || iters == 0) {
    int imageSizes[5] = {128, 512, 4096, 8192, 16384};
    int iterSizes[5] = {5, 1, 15, 20, 40};
    imageSize = imageSizes[op.getOptionInt("size") - 1];
    speckleSize = imageSize / 2;
    iters = iterSizes[op.getOptionInt("size") - 1];
  }

  // create timing events
  checkCudaErrors(hipEventCreate(&start));
  checkCudaErrors(hipEventCreate(&stop));

  if (!quiet) {
      printf("WG size of kernel = %d X %d\n", BLOCK_SIZE, BLOCK_SIZE);
      printf("Image Size: %d x %d\n", imageSize, imageSize);
      printf("Speckle size: %d x %d\n", speckleSize, speckleSize);
      printf("Num Iterations: %d\n\n", iters);
  }

  // run workload
  int passes = op.getOptionInt("passes");
  for (int i = 0; i < passes; i++) {
    float *matrix = NULL;
    if (uvm || uvm_advise || uvm_prefetch || uvm_prefetch_advise || zero_copy || pud) {
        checkCudaErrors(hipMallocManaged(&matrix, imageSize * imageSize * sizeof(float)));
    } else if (copy) {
        checkCudaErrors(hipHostMalloc(&matrix, imageSize * imageSize * sizeof(float)));
        assert(matrix);
    } else if (pageable) {
        matrix = (float*)malloc(imageSize * imageSize * sizeof(float));
        assert(matrix);
    }
    random_matrix(matrix, imageSize, imageSize);
    if (!quiet) {
        printf("Pass %d:\n", i);
    }
    // (taeklim)
    if (uvm) {
    } else if (zero_copy) {
        checkCudaErrors(hipMemAdvise(matrix, sizeof(float) * imageSize * imageSize, hipMemAdviseSetAccessedBy, 0));
    } else if (pud) {
        checkCudaErrors(hipMemAdvise(matrix, sizeof(float) * imageSize * imageSize, hipMemAdviseSetAccessedBy, 0));
    }

    float time = srad(resultDB, op, matrix, imageSize, speckleSize, iters, ofile, sem);
    if (!quiet) {
        printf("Running SRAD...Done.\n");
    }
    if (coop) {
        // if using cooperative groups, add result to compare the 2 times
        char atts[1024];
        sprintf(atts, "img:%d,speckle:%d,iter:%d", imageSize, speckleSize, iters);
        float time_gridsync = srad_gridsync(resultDB, op, matrix, imageSize, speckleSize, iters);
        if(!quiet) {
            if(time_gridsync == FLT_MAX) {
                printf("Running SRAD with cooperative groups...Failed.\n");
            } else {
                printf("Running SRAD with cooperative groups...Done.\n");
            }
        }
        if(time_gridsync == FLT_MAX) {
            resultDB.AddResult("srad_gridsync_speedup", atts, "N", time/time_gridsync);
        }
    }
    if (uvm || uvm_advise || uvm_prefetch || uvm_prefetch_advise || zero_copy || pud) {
        checkCudaErrors(hipFree(matrix));
    } else if (copy) {
        checkCudaErrors(hipHostFree(matrix));
    } else if (pageable) {
        free(matrix);
    }
  }
}

////////////////////////////////////////////////////////////////////////////////////////////////////
/// <summary>	Srads. </summary>
///
/// <remarks>	Ed, 5/20/2020. </remarks>
///
/// <param name="resultDB">   	[in,out] The result database. </param>
/// <param name="op">		  	[in,out] The operation. </param>
/// <param name="matrix">	  	[in,out] If non-null, the matrix. </param>
/// <param name="imageSize">  	Size of the image. </param>
/// <param name="speckleSize">	Size of the speckle. </param>
/// <param name="iters">	  	The iters. </param>
///
/// <returns>	A float. </returns>
////////////////////////////////////////////////////////////////////////////////////////////////////

float srad(ResultDatabase &resultDB, OptionParser &op, float* matrix, int imageSize,
        int speckleSize, int iters, ofstream &ofile, sem_t *sem) {
    const bool uvm = op.getOptionBool("uvm");
    const bool zero_copy = op.getOptionBool("zero-copy");
    const bool uvm_advise = op.getOptionBool("uvm-advise");
    const bool uvm_prefetch = op.getOptionBool("uvm-prefetch");
    const bool uvm_prefetch_advise = op.getOptionBool("uvm-prefetch-advise");
    const bool coop = op.getOptionBool("coop");
    const bool copy = op.getOptionBool("copy");
    const bool pageable = op.getOptionBool("pageable");
    const bool pud = op.getOptionBool("pud");
    const bool is_barrier = op.getOptionBool("sem");
    string bench_name = op.getOptionString("bench");
    int device = 0;
    checkCudaErrors(hipGetDevice(&device));

    kernelTime = 0.0f;
    transferTime = 0.0f;
    int rows, cols, size_I, size_R, niter, iter;
    float *I, *J, lambda, q0sqr, sum, sum2, tmp, meanROI, varROI;

    float *J_cuda;
    float *C_cuda;
    float *E_C, *W_C, *N_C, *S_C;

    unsigned int r1, r2, c1, c2;
    float *c;

    rows = imageSize;  // number of rows in the domain
    cols = imageSize;  // number of cols in the domain
    if ((rows % 16 != 0) || (cols % 16 != 0)) {
        fprintf(stderr, "rows and cols must be multiples of 16\n");
        exit(1);
    }
    r1 = 0;            // y1 position of the speckle
    r2 = speckleSize;  // y2 position of the speckle
    c1 = 0;            // x1 position of the speckle
    c2 = speckleSize;  // x2 position of the speckle
    lambda = 0.5;      // Lambda value
    niter = iters;     // number of iterations

    size_I = cols * rows;
    size_R = (r2 - r1 + 1) * (c2 - c1 + 1);

    if (uvm || uvm_advise || uvm_prefetch || uvm_prefetch_advise || zero_copy || pud) {
        checkCudaErrors(hipMallocManaged(&J, sizeof(float) * size_I));
        checkCudaErrors(hipMallocManaged(&c, sizeof(float) * size_I));
    } else if (copy) {
        //checkCudaErrors(hipHostMalloc(&I, size_I * sizeof(float)));
        checkCudaErrors(hipHostMalloc(&J, size_I * sizeof(float)));
        checkCudaErrors(hipHostMalloc(&c, size_I * sizeof(float)));
    } else if (pageable) {
        I = (float *)malloc(size_I * sizeof(float));
        assert(I);
        J = (float *)malloc(size_I * sizeof(float));
        assert(J);
        c = (float *)malloc(sizeof(float) * size_I);
        assert(c);
    }

    // Allocate device memory
    if (uvm || uvm_advise || uvm_prefetch || uvm_prefetch_advise || zero_copy || pud) {
        J_cuda = J;
        C_cuda = c;
        checkCudaErrors(hipMallocManaged((void **)&E_C, sizeof(float) * size_I));
        printf("E_C:%ld\n", E_C);
        checkCudaErrors(hipMallocManaged((void **)&W_C, sizeof(float) * size_I));
        printf("W_C:%ld\n", W_C);
        checkCudaErrors(hipMallocManaged((void **)&S_C, sizeof(float) * size_I));
        printf("S_C:%ld\n", S_C);
        checkCudaErrors(hipMallocManaged((void **)&N_C, sizeof(float) * size_I));
        printf("N_C:%ld\n", N_C);
    } else if (copy || pageable) {
        checkCudaErrors(hipMalloc((void **)&J_cuda, sizeof(float) * size_I));
        checkCudaErrors(hipMalloc((void **)&C_cuda, sizeof(float) * size_I));
        checkCudaErrors(hipMalloc((void **)&E_C, sizeof(float) * size_I));
        checkCudaErrors(hipMalloc((void **)&W_C, sizeof(float) * size_I));
        checkCudaErrors(hipMalloc((void **)&S_C, sizeof(float) * size_I));
        checkCudaErrors(hipMalloc((void **)&N_C, sizeof(float) * size_I));
    }

    // copy random matrix
    if (uvm || uvm_advise || uvm_prefetch || uvm_prefetch_advise || copy || zero_copy || pud) {
        I = matrix;
    } else if (pageable) {
        memcpy(I, matrix, rows*cols*sizeof(float));
    }

    for (int k = 0; k < size_I; k++) {
        J[k] = (float)exp(I[k]);
    }
    for (iter = 0; iter < niter; iter++) {
        sum = 0;
        sum2 = 0;
        for (int i = r1; i <= r2; i++) {
            for (int j = c1; j <= c2; j++) {
                tmp = J[i * cols + j];
                sum += tmp;
                sum2 += tmp * tmp;
            }
        }
        meanROI = sum / size_R;
        varROI = (sum2 / size_R) - meanROI * meanROI;
        q0sqr = varROI / (meanROI * meanROI);

        // Currently the input size must be divided by 16 - the block size
        int block_x = cols / BLOCK_SIZE;
        int block_y = rows / BLOCK_SIZE;

        dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
        dim3 dimGrid(block_x, block_y);

        // Copy data from main memory to device memory
        if (!copy && !pageable) {
            checkCudaErrors(hipEventRecord(start, 0));
        }
        if (uvm) {
            // do nothing
        } else if (zero_copy) {
            checkCudaErrors(hipMemAdvise(J_cuda, sizeof(float) * size_I, hipMemAdviseSetAccessedBy, 0));
            checkCudaErrors(hipMemAdvise(C_cuda, sizeof(float) * size_I, hipMemAdviseSetAccessedBy, 0));

            checkCudaErrors(hipMemAdvise(E_C, sizeof(float) * size_I, hipMemAdviseSetAccessedBy, 0));
            checkCudaErrors(hipMemAdvise(W_C, sizeof(float) * size_I, hipMemAdviseSetAccessedBy, 0));
            checkCudaErrors(hipMemAdvise(S_C, sizeof(float) * size_I, hipMemAdviseSetAccessedBy, 0));
            checkCudaErrors(hipMemAdvise(N_C, sizeof(float) * size_I, hipMemAdviseSetAccessedBy, 0));
        } else if (pud) {
//            checkCudaErrors(hipMemAdvise(E_C, sizeof(float) * size_I, hipMemAdviseSetAccessedBy, 0));
//            checkCudaErrors(hipMemAdvise(W_C, sizeof(float) * size_I, hipMemAdviseSetAccessedBy, 0));
//            checkCudaErrors(hipMemAdvise(S_C, sizeof(float) * size_I, hipMemAdviseSetAccessedBy, 0));
//            checkCudaErrors(hipMemAdvise(N_C, sizeof(float) * size_I, hipMemAdviseSetAccessedBy, 0));
        } else if (uvm_advise) {
            checkCudaErrors(hipMemAdvise(J_cuda, sizeof(float) * size_I, hipMemAdviseSetPreferredLocation, device));
        } else if (uvm_prefetch) {
            checkCudaErrors(hipMemPrefetchAsync(J_cuda, sizeof(float) * size_I, device));
        } else if (uvm_prefetch_advise) {
            checkCudaErrors(hipMemAdvise(J_cuda, sizeof(float) * size_I, hipMemAdviseSetPreferredLocation, device));
            checkCudaErrors(hipMemPrefetchAsync(J_cuda, sizeof(float) * size_I, device));
        } else if (copy || pageable) {
            if (is_barrier && pageable) {
                int sval;
                sem_post(sem);
                sem_getvalue(sem, &sval);
                while (sval == 1) {
                    sem_getvalue(sem, &sval);
                }
                printf("[Barrier] Copying starts\n");
            }
            checkCudaErrors(hipEventRecord(start, 0));
            checkCudaErrors(hipMemcpy(J_cuda, J, sizeof(float) * size_I, hipMemcpyHostToDevice));
        }
        checkCudaErrors(hipEventRecord(stop, 0));
        checkCudaErrors(hipEventSynchronize(stop));
        checkCudaErrors(hipEventElapsedTime(&elapsed, start, stop));
        transferTime += elapsed * 1.e-3;

        // (taeklim): Waiting for the other apps finishes the initialization
        if (is_barrier && uvm) {
            int sval;
            sem_post(sem);
            sem_getvalue(sem, &sval);
            while (sval == 1) {
                sem_getvalue(sem, &sval);
            }
            printf("[Barrier] Kernel starts\n");
        }

        // Run kernels
        checkCudaErrors(hipEventRecord(start, 0));
        srad_cuda_1<<<dimGrid, dimBlock>>>(E_C, W_C, N_C, S_C, J_cuda, C_cuda, cols,
                rows, q0sqr);
        checkCudaErrors(hipEventRecord(stop, 0));
        checkCudaErrors(hipEventSynchronize(stop));
        checkCudaErrors(hipEventElapsedTime(&elapsed, start, stop));
        kernelTime += elapsed * 1.e-3;
        CHECK_CUDA_ERROR();

        checkCudaErrors(hipEventRecord(start, 0));
        srad_cuda_2<<<dimGrid, dimBlock>>>(E_C, W_C, N_C, S_C, J_cuda, C_cuda, cols,
                rows, lambda, q0sqr);
        checkCudaErrors(hipEventRecord(stop, 0));
        checkCudaErrors(hipEventSynchronize(stop));
        checkCudaErrors(hipEventElapsedTime(&elapsed, start, stop));
        kernelTime += elapsed * 1.e-3;
        CHECK_CUDA_ERROR();

        // Copy data from device memory to main memory
        checkCudaErrors(hipEventRecord(start, 0));

        if (uvm || zero_copy) {
            // do nothing
        } else if (uvm_advise) {
            checkCudaErrors(hipMemAdvise(J_cuda, sizeof(float) * size_I, hipMemAdviseSetPreferredLocation, hipCpuDeviceId));
        } else if (uvm_prefetch) {
            checkCudaErrors(hipMemPrefetchAsync(J_cuda, sizeof(float) * size_I, hipCpuDeviceId));
        } else if (uvm_prefetch_advise) {
            checkCudaErrors(hipMemAdvise(J_cuda, sizeof(float) * size_I, hipMemAdviseSetPreferredLocation, hipCpuDeviceId));
            checkCudaErrors(hipMemAdvise(J_cuda, sizeof(float) * size_I, hipMemAdviseSetReadMostly, hipCpuDeviceId));
            checkCudaErrors(hipMemPrefetchAsync(J_cuda, sizeof(float) * size_I, hipCpuDeviceId));
        } else if (pageable || copy) {
            checkCudaErrors(hipMemcpy(J, J_cuda, sizeof(float) * size_I, hipMemcpyDeviceToHost));
        }
        checkCudaErrors(hipEventRecord(stop, 0));
        checkCudaErrors(hipEventSynchronize(stop));
        checkCudaErrors(hipEventElapsedTime(&elapsed, start, stop));
        transferTime += elapsed * 1.e-3;
    }

    char atts[1024];
    sprintf(atts, "img:%d,speckle:%d,iter:%d", imageSize, speckleSize, iters);
    resultDB.AddResult("srad_kernel_time", atts, "sec", kernelTime);
    resultDB.AddResult("srad_transfer_time", atts, "sec", transferTime);
    resultDB.AddResult("srad_total_time", atts, "sec", kernelTime + transferTime);
    resultDB.AddResult("srad_parity", atts, "N", transferTime / kernelTime);
    resultDB.AddOverall("Time", "sec", kernelTime+transferTime);
    ofile << bench_name << ", " << kernelTime + transferTime << ", " << endl;

//    string outfile = op.getOptionString("outputFile");
//    if (!outfile.empty()) {
//        // Printing output
//        if (!op.getOptionBool("quiet")) {
//            printf("Writing output to %s\n", outfile.c_str());
//        }
//        FILE *fp = NULL;
//        fp = fopen(outfile.c_str(), "w");
//        if (!fp) {
//            printf("Error: Unable to write to file %s\n", outfile.c_str());
//        } else {
//            for (int i = 0; i < rows; i++) {
//                for (int j = 0; j < cols; j++) {
//                    fprintf(fp, "%.5f ", J[i * cols + j]);
//                }
//                fprintf(fp, "\n");
//            }
//            fclose(fp);
//        }
//    }
    // write results to validate with srad_gridsync
    check = (float*) malloc(sizeof(float) * size_I);
    assert(check);
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            check[i*cols+j] = J[i*cols+j];
        }
    }

    if (uvm || uvm_advise || uvm_prefetch || uvm_prefetch_advise || zero_copy) {
        checkCudaErrors(hipFree(C_cuda));
        checkCudaErrors(hipFree(J_cuda));
        checkCudaErrors(hipFree(E_C));
        checkCudaErrors(hipFree(W_C));
        checkCudaErrors(hipFree(N_C));
        checkCudaErrors(hipFree(S_C));
    } else if (copy) {
        //hipHostFree(I);
        hipHostFree(J);
        hipHostFree(c);
        checkCudaErrors(hipFree(C_cuda));
        checkCudaErrors(hipFree(J_cuda));
        checkCudaErrors(hipFree(E_C));
        checkCudaErrors(hipFree(W_C));
        checkCudaErrors(hipFree(N_C));
        checkCudaErrors(hipFree(S_C));
    } else if (pageable) {
        free(I);
        free(J);
        free(c);
        checkCudaErrors(hipFree(C_cuda));
        checkCudaErrors(hipFree(J_cuda));
        checkCudaErrors(hipFree(E_C));
        checkCudaErrors(hipFree(W_C));
        checkCudaErrors(hipFree(N_C));
        checkCudaErrors(hipFree(S_C));
    }
    return kernelTime;
}

////////////////////////////////////////////////////////////////////////////////////////////////////
/// <summary>	Srad gridsync with UVM and gridsync. </summary>
///
/// <remarks>	Ed, 5/20/2020. </remarks>
///
/// <param name="resultDB">   	[in,out] The result database. </param>
/// <param name="op">		  	[in,out] The operation. </param>
/// <param name="matrix">	  	[in,out] If non-null, the matrix. </param>
/// <param name="imageSize">  	Size of the image. </param>
/// <param name="speckleSize">	Size of the speckle. </param>
/// <param name="iters">	  	The iters. </param>
///
/// <returns>	A float. </returns>
////////////////////////////////////////////////////////////////////////////////////////////////////

float srad_gridsync(ResultDatabase &resultDB, OptionParser &op, float* matrix, int imageSize, int speckleSize, int iters) {
    const bool uvm = op.getOptionBool("uvm");
    const bool copy = op.getOptionBool("copy");
    const bool pageable = op.getOptionBool("pageable");
    const bool uvm_advise = op.getOptionBool("uvm-advise");
    const bool uvm_prefetch = op.getOptionBool("uvm-prefetch");
    const bool uvm_prefetch_advise = op.getOptionBool("uvm-prefetch-advise");
    const bool coop = op.getOptionBool("coop");
    int device = 0;
    checkCudaErrors(hipGetDevice(&device));
    
    kernelTime = 0.0f;
    transferTime = 0.0f;
    int rows, cols, size_I, size_R, niter, iter;
    float *I, *J, lambda, q0sqr, sum, sum2, tmp, meanROI, varROI;

  float *J_cuda;
  float *C_cuda;
  float *E_C, *W_C, *N_C, *S_C;

  unsigned int r1, r2, c1, c2;
  float *c;

  rows = imageSize;  // number of rows in the domain
  cols = imageSize;  // number of cols in the domain
  if ((rows % 16 != 0) || (cols % 16 != 0)) {
    fprintf(stderr, "rows and cols must be multiples of 16\n");
    exit(1);
  }
  r1 = 0;            // y1 position of the speckle
  r2 = speckleSize;  // y2 position of the speckle
  c1 = 0;            // x1 position of the speckle
  c2 = speckleSize;  // x2 position of the speckle
  lambda = 0.5;      // Lambda value
  niter = iters;     // number of iterations

  size_I = cols * rows;
  size_R = (r2 - r1 + 1) * (c2 - c1 + 1);

  if (uvm || uvm_advise || uvm_prefetch || uvm_prefetch_advise) {
    checkCudaErrors(hipMallocManaged((void **)&J, sizeof(float) * size_I));
    checkCudaErrors(hipMallocManaged((void **)&c, sizeof(float) * size_I));
  } else if (copy) {
    checkCudaErrors(hipHostMalloc((void **)&J, sizeof(float) * size_I));
    checkCudaErrors(hipHostMalloc((void **)&c, sizeof(float) * size_I));
  } else if (pageable) {
    I = (float *)malloc(size_I * sizeof(float));
    assert(I);
    J = (float *)malloc(size_I * sizeof(float));
    assert(J);
    c = (float *)malloc(sizeof(float) * size_I);
    assert(c);
  }

  // Allocate device memory
  if (uvm || uvm_advise || uvm_prefetch || uvm_prefetch_advise) {
    J_cuda = J;
    C_cuda = c;
    checkCudaErrors(hipMallocManaged((void **)&E_C, sizeof(float) * size_I));
    checkCudaErrors(hipMallocManaged((void **)&W_C, sizeof(float) * size_I));
    checkCudaErrors(hipMallocManaged((void **)&S_C, sizeof(float) * size_I));
    checkCudaErrors(hipMallocManaged((void **)&N_C, sizeof(float) * size_I));
  } else if (copy || pageable) {
    checkCudaErrors(hipMalloc((void **)&J_cuda, sizeof(float) * size_I));
    checkCudaErrors(hipMalloc((void **)&C_cuda, sizeof(float) * size_I));
    checkCudaErrors(hipMalloc((void **)&E_C, sizeof(float) * size_I));
    checkCudaErrors(hipMalloc((void **)&W_C, sizeof(float) * size_I));
    checkCudaErrors(hipMalloc((void **)&S_C, sizeof(float) * size_I));
    checkCudaErrors(hipMalloc((void **)&N_C, sizeof(float) * size_I));
  }

  // Generate a random matrix
  if (uvm || uvm_advise || uvm_prefetch || uvm_prefetch_advise) {
    I = matrix;
  } else if (pageable || copy) {
    memcpy(I, matrix, rows*cols*sizeof(float));
  }

  for (int k = 0; k < size_I; k++) {
    J[k] = (float)exp(I[k]);
  }
  for (iter = 0; iter < niter; iter++) {
    sum = 0;
    sum2 = 0;
    for (int i = r1; i <= r2; i++) {
      for (int j = c1; j <= c2; j++) {
        tmp = J[i * cols + j];
        sum += tmp;
        sum2 += tmp * tmp;
      }
    }
    meanROI = sum / size_R;
    varROI = (sum2 / size_R) - meanROI * meanROI;
    q0sqr = varROI / (meanROI * meanROI);

    // Currently the input size must be divided by 16 - the block size
    int block_x = cols / BLOCK_SIZE;
    int block_y = rows / BLOCK_SIZE;

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(block_x, block_y);

    // Copy data from main memory to device memory
    if (!pageable && !copy) {
        checkCudaErrors(hipEventRecord(start, 0));
    }
    // timing incorrect for page fault
    // J_cuda = J;
    // C_cuda = c;
    if (uvm) {
      // do nothing
    } else if (uvm_advise) {
      checkCudaErrors(hipMemAdvise(J_cuda, sizeof(float) * size_I, hipMemAdviseSetPreferredLocation, device));
    } else if (uvm_prefetch) {
      checkCudaErrors(hipMemPrefetchAsync(J_cuda, sizeof(float) * size_I, device));
    } else if (uvm_prefetch_advise) {
      checkCudaErrors(hipMemAdvise(J_cuda, sizeof(float) * size_I, hipMemAdviseSetPreferredLocation, device));
      checkCudaErrors(hipMemPrefetchAsync(J_cuda, sizeof(float) * size_I, device));
    } else if (copy || pageable) {
        checkCudaErrors(hipEventRecord(start, 0));
        checkCudaErrors(hipMemcpy(J_cuda, J, sizeof(float) * size_I, hipMemcpyHostToDevice));
    }
    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&elapsed, start, stop));
    transferTime += elapsed * 1.e-3;

    // Create srad_params struct
    srad_params params;
    params.E_C = E_C;
    params.W_C = W_C;
    params.N_C = N_C;
    params.S_C = S_C;
    params.J_cuda = J_cuda;
    params.C_cuda = C_cuda;
    params.cols = cols;
    params.rows = rows;
    params.lambda = lambda;
    params.q0sqr = q0sqr;
    void* p_params = {&params};

    // Run kernels
    checkCudaErrors(hipEventRecord(start, 0));
    checkCudaErrors(hipLaunchCooperativeKernel((void*)srad_cuda_3, dimGrid, dimBlock, &p_params));
    //srad_cuda_3<<<dimGrid, dimBlock>>>(E_C, W_C, N_C, S_C, J_cuda, C_cuda, cols,
                                       //rows, lambda, q0sqr);
    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&elapsed, start, stop));
    kernelTime += elapsed * 1.e-3;
    hipError_t err = hipGetLastError();                                     
    if (err != hipSuccess)                                                   
    {                                                                         
      printf("error=%d name=%s at "                                         
               "ln: %d\n  ",err,hipGetErrorString(err),__LINE__);            
      if (uvm || uvm_advise || uvm_prefetch || uvm_prefetch_advise) {
        checkCudaErrors(hipFree(C_cuda));
        checkCudaErrors(hipFree(J_cuda));
        checkCudaErrors(hipFree(E_C));
        checkCudaErrors(hipFree(W_C));
        checkCudaErrors(hipFree(N_C));
        checkCudaErrors(hipFree(S_C));
      }
      else if (pageable) {
        checkCudaErrors(hipFree(C_cuda));
        checkCudaErrors(hipFree(J_cuda));
        checkCudaErrors(hipFree(E_C));
        checkCudaErrors(hipFree(W_C));
        checkCudaErrors(hipFree(N_C));
        checkCudaErrors(hipFree(S_C));

        free(I);
        free(J);
        free(c);
      } else if (copy) {
        checkCudaErrors(hipFree(C_cuda));
        checkCudaErrors(hipFree(J_cuda));
        checkCudaErrors(hipFree(E_C));
        checkCudaErrors(hipFree(W_C));
        checkCudaErrors(hipFree(N_C));
        checkCudaErrors(hipFree(S_C));
        //hipHostFree(I);
        hipHostFree(J);
        hipHostFree(c);
      }
    return FLT_MAX;
    }                                                                     

    // Copy data from device memory to main memory
    checkCudaErrors(hipEventRecord(start, 0));
    if (uvm) {
      // do nothing
    } else if (uvm_advise) {
      checkCudaErrors(hipMemAdvise(J, sizeof(float) * size_I, hipMemAdviseSetPreferredLocation, hipCpuDeviceId));
      checkCudaErrors(hipMemAdvise(J, sizeof(float) * size_I, hipMemAdviseSetReadMostly, hipCpuDeviceId));
    } else if (uvm_prefetch) {
      checkCudaErrors(hipMemPrefetchAsync(J, sizeof(float) * size_I, hipCpuDeviceId));
    } else if (uvm_prefetch_advise) {
      checkCudaErrors(hipMemAdvise(J, sizeof(float) * size_I, hipMemAdviseSetPreferredLocation, hipCpuDeviceId));
      checkCudaErrors(hipMemAdvise(J, sizeof(float) * size_I, hipMemAdviseSetReadMostly, hipCpuDeviceId));
      checkCudaErrors(hipMemPrefetchAsync(J, sizeof(float) * size_I, hipCpuDeviceId));
    } else if (pageable || copy) {
      checkCudaErrors(hipMemcpy(J, J_cuda, sizeof(float) * size_I, hipMemcpyDeviceToHost));
    }
    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&elapsed, start, stop));
    transferTime += elapsed * 1.e-3;
  }

    char atts[1024];
    sprintf(atts, "img:%d,speckle:%d,iter:%d", imageSize, speckleSize, iters);
    resultDB.AddResult("srad_gridsync_kernel_time", atts, "sec", kernelTime);
    resultDB.AddResult("srad_gridsync_transer_time", atts, "sec", transferTime);
    resultDB.AddResult("srad_gridsync_total_time", atts, "sec", kernelTime + transferTime);
    resultDB.AddResult("srad_gridsync_parity", atts, "N", transferTime / kernelTime);

  // validate result with result obtained by gridsync
  for (int i = 0; i < rows; i++) {
      for (int j = 0; j < cols; j++) {
          if(check[i*cols+j] - J[i*cols+j] > 0.0001) {
              // known bug: with and without gridsync have 10e-5 difference in row 16
              //printf("Error: Validation failed at row %d, col %d\n", i, j);
              //return FLT_MAX;
          }
      }
  }
  if (uvm || uvm_advise || uvm_prefetch || uvm_prefetch_advise) {
    CUDA_SAFE_CALL(hipFree(C_cuda));
    CUDA_SAFE_CALL(hipFree(J_cuda));
    CUDA_SAFE_CALL(hipFree(E_C));
    CUDA_SAFE_CALL(hipFree(W_C));
    CUDA_SAFE_CALL(hipFree(N_C));
    CUDA_SAFE_CALL(hipFree(S_C));
  } else if (pageable) {
    free(I);
    free(J);
    free(c);
    CUDA_SAFE_CALL(hipFree(C_cuda));
    CUDA_SAFE_CALL(hipFree(J_cuda));
    CUDA_SAFE_CALL(hipFree(E_C));
    CUDA_SAFE_CALL(hipFree(W_C));
    CUDA_SAFE_CALL(hipFree(N_C));
    CUDA_SAFE_CALL(hipFree(S_C));
  } else if (copy) {
    hipHostFree(I);
    hipHostFree(J);
    hipHostFree(c);
    CUDA_SAFE_CALL(hipFree(C_cuda));
    CUDA_SAFE_CALL(hipFree(J_cuda));
    CUDA_SAFE_CALL(hipFree(E_C));
    CUDA_SAFE_CALL(hipFree(W_C));
    CUDA_SAFE_CALL(hipFree(N_C));
    CUDA_SAFE_CALL(hipFree(S_C));
  }
  return kernelTime;
}

////////////////////////////////////////////////////////////////////////////////////////////////////
/// <summary>	Random matrix. </summary>
///
/// <remarks>	Ed, 5/20/2020. </remarks>
///
/// <param name="I">   	[in,out] If non-null, zero-based index of the. </param>
/// <param name="rows">	The rows. </param>
/// <param name="cols">	The cols. </param>
////////////////////////////////////////////////////////////////////////////////////////////////////

void random_matrix(float *I, int rows, int cols) {
  for (int i = 0; i < rows; i++) {
    for (int j = 0; j < cols; j++) {
      I[i * cols + j] = rand() / (float)RAND_MAX;
    }
  }
}

