////////////////////////////////////////////////////////////////////////////////////////////////////
// file:	altis\src\cuda\common\main.cpp
//
// summary:	Implements the main class
// 
// origin: SHOC (https://github.com/vetter/shoc)
////////////////////////////////////////////////////////////////////////////////////////////////////

#include <iostream>
#include <cstdlib>
#include <fstream>

#include <hip/hip_runtime.h> 
#include <hip/hip_runtime.h>


#include "ResultDatabase.h"
#include "OptionParser.h"
#include "Utility.h"
#include "cudacommon.h"

#include <unistd.h>
#define BYTE_PER_MB 1048576ULL
#define MAX_MEM_SIZE 40545

using namespace std;

// Forward Declarations
void addBenchmarkSpecOptions(OptionParser &op);
void RunBenchmark(ResultDatabase &resultDB, OptionParser &op, ofstream &ofile, sem_t *sem);
//void RunBenchmarkSem(ResultDatabase &resultDB, OptionParser &op, ofstream &ofile, sem_t *sem);

// ****************************************************************************
// Function: EnumerateDevicesAndChoose
//
// Purpose:
//   This function queries cuda about the available gpus in the system, prints
//   those results to standard out, and selects a device for use in the
//   benchmark.
//
// Arguments:
//   chooseDevice: logical number for the desired device
//   properties: whether or not to print device properties and exit
//
// Returns:  nothing
//
// Programmer: Jeremy Meredith
// Creation:
//
// Modifications:
//   Jeremy Meredith, Tue Oct  9 17:27:04 EDT 2012
//   Added a windows-specific --noprompt, which unless the user passes it,
//   prompts the user to press enter before the program exits on Windows.
//   This is because on Windows, the console disappears when the program
//   exits, but our results go to the console.
//
// ****************************************************************************
void EnumerateDevicesAndChoose(int chooseDevice, bool properties, bool quiet)
{
    hipSetDevice(chooseDevice);
    int actualdevice;
    hipGetDevice(&actualdevice);

    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (properties)
    {
        cout << "Number of devices = " << deviceCount << "\n";
    }
    string deviceName = "";
    for (int device = 0; device < deviceCount; ++device)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, device);
        if (device == actualdevice)
            deviceName = deviceProp.name;
        if (properties)
        {
            cout << "Device " << device << ":\n";
            cout << "  name               = '" << deviceProp.name << "'"
                    << endl;
            cout << "  totalGlobalMem     = " << HumanReadable(
                    deviceProp.totalGlobalMem) << endl;
            cout << "  sharedMemPerBlock  = " << HumanReadable(
                    deviceProp.sharedMemPerBlock) << endl;
            cout << "  regsPerBlock       = " << deviceProp.regsPerBlock
                    << endl;
            cout << "  warpSize           = " << deviceProp.warpSize << endl;
            cout << "  memPitch           = " << HumanReadable(
                    deviceProp.memPitch) << endl;
            cout << "  maxThreadsPerBlock = " << deviceProp.maxThreadsPerBlock
                    << endl;
            cout << "  maxThreadsDim[3]   = " << deviceProp.maxThreadsDim[0]
                    << "," << deviceProp.maxThreadsDim[1] << ","
                    << deviceProp.maxThreadsDim[2] << endl;
            cout << "  maxGridSize[3]     = " << deviceProp.maxGridSize[0]
                    << "," << deviceProp.maxGridSize[1] << ","
                    << deviceProp.maxGridSize[2] << endl;
            cout << "  totalConstMem      = " << HumanReadable(
                    deviceProp.totalConstMem) << endl;
            cout << "  major (hw version) = " << deviceProp.major << endl;
            cout << "  minor (hw version) = " << deviceProp.minor << endl;
            cout << "  clockRate          = " << deviceProp.clockRate << endl;
            cout << "  textureAlignment   = " << deviceProp.textureAlignment
                    << endl;
        }
    }
    if(properties) {
        return;
    }
    if(!quiet) {
        cout << "Chose device:"
            << " name='"<<deviceName<<"'"
            << " index="<<actualdevice
            << endl;
    }
}

void checkCudaFeatureAvailability(OptionParser &op) {
    int device = 0;
    checkCudaErrors(hipGetDevice(&device));
    hipDeviceProp_t deviceProp;
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, device));
    
    // Check UVM availability
    if (op.getOptionBool("uvm") || op.getOptionBool("uvm-advise") ||
            op.getOptionBool("uvm-prefetch") || op.getOptionBool("uvm-prefetch-advise")) {
        if (!deviceProp.unifiedAddressing) {
            std::cerr << "device doesn't support unified addressing, exiting..." << std::endl;
            safe_exit(-1);
        }
    }

    // Check Cooperative Group availability
    if (op.getOptionBool("coop")) {
        if (!deviceProp.cooperativeLaunch) {
            std::cerr << "device doesn't support cooperative kernels, exiting..." << std::endl;
            safe_exit(-1);
        }
    }

    // Check Dynamic Parallelism availability
    if (op.getOptionBool("dyn")) {
        int runtimeVersion = 0;
        checkCudaErrors(hipRuntimeGetVersion(&runtimeVersion));
        if (runtimeVersion < 5000) {
            std::cerr << "CUDA runtime version less than 5.0, doesn't support \
                dynamic parallelism, exiting..." << std::endl;
            safe_exit(-1);
        }
    }

    // Check CUDA Graphs availability
    if (op.getOptionBool("graph")) {
        int runtimeVersion = 0;
        checkCudaErrors(hipRuntimeGetVersion(&runtimeVersion));
        if (runtimeVersion < 10000) {
            std::cerr << "CUDA runtime version less than 10.0, doesn't support \
                CUDA Graph, exiting..." << std::endl;
            safe_exit(-1);
        }
    }
}

// ****************************************************************************
// Function: main
//
// Purpose:
//   The main function takes care of initialization (device and MPI),  then
//   performs the benchmark and prints results.
//
// Arguments:
//
//
// Programmer: Jeremy Meredith
// Creation:
//
// Modifications:
//   Jeremy Meredith, Wed Nov 10 14:20:47 EST 2010
//   Split timing reports into detailed and summary.  For serial code, we
//   report all trial values, and for parallel, skip the per-process vals.
//   Also detect and print outliers from parallel runs.
//
// ****************************************************************************
int main(int argc, char *argv[])
{

    int ret = 0;

    try
    {
        // Get args
        OptionParser op;

        // Add shared options to the parser
        op.addOption("properties", OPT_BOOL, "0",
                "show properties for available platforms and devices", 'p');
        op.addOption("device", OPT_VECINT, "0",
                "specify device(s) to run on", 'd');
        op.addOption("passes", OPT_INT, "10", "specify number of passes", 'n');
        op.addOption("size", OPT_INT, "1", "specify problem size", 's');
        op.addOption("verbose", OPT_BOOL, "0", "enable verbose output", 'v');
        op.addOption("quiet", OPT_BOOL, "0", "enable concise output", 'q');
        op.addOption("configFile", OPT_STRING, "", "path of configuration file", 'c');
        op.addOption("inputFile", OPT_STRING, "", "path of input file", 'i');
        op.addOption("outputFile", OPT_STRING, "", "path of output file", 'o');
        op.addOption("metricsFile", OPT_STRING, "", "path of file to write metrics to", 'm');
        // (taeklim): Add dummy memory allocation to limit memory size
        op.addOption("dummy", OPT_INT, "0", "size for allocating dummy memory in MB");
        op.addOption("oversub-frac", OPT_FLOAT, "0", "fraction for memory oversubscription");

        // Add options for turn on/off CUDA features
        // (taeklim)
        op.addOption("pageable", OPT_BOOL, "0", "enable pageable memory allocation (hipMalloc)");
        op.addOption("async", OPT_BOOL, "0", "enable asynchronous copy (hipMemcpyAsync)");
        op.addOption("copy", OPT_BOOL, "0", "enable pinned memory allocation (hipHostMalloc)");
        op.addOption("dha", OPT_BOOL, "0", "enable direct-host-access (hipHostMalloc w/o hipMemcpy)");
        op.addOption("uvm", OPT_BOOL, "0", "enable CUDA Unified Virtual Memory, only demand paging");
        op.addOption("uvm-copy", OPT_BOOL, "0", "enable CUDA Unified Virtual Memory with manual memcopy");
        op.addOption("uvm-oversub", OPT_BOOL, "0", "enable CUDA Unified Virtual Memory with oversubscription");

        op.addOption("uvm-advise", OPT_BOOL, "0", "guide the driver about memory usage patterns");
        op.addOption("uvm-prefetch", OPT_BOOL, "0", "prefetch memory the specified destination device");
        op.addOption("uvm-prefetch-advise", OPT_BOOL, "0", "prefetch memory the specified destination device with memory guidance on");
        op.addOption("zero-copy", OPT_BOOL, "0", "enable CUDA Unified Virtual Memory with zero-copy");
        op.addOption("pud", OPT_BOOL, "0", "enable Gpuddle");
        op.addOption("emoji", OPT_BOOL, "0", "enable CUDA Unified Virtual Memory with zero-copy");
        op.addOption("coal", OPT_BOOL, "0", "enable warp coalescing");

        op.addOption("coop", OPT_BOOL, "0", "enable CUDA Cooperative Groups");
        op.addOption("dyn", OPT_BOOL, "0", "enable CUDA Dynamic Parallelism");
        op.addOption("graph", OPT_BOOL, "0", "enable CUDA Graphs");
        // (taeklim)
        op.addOption("sem", OPT_BOOL, "0", "enable barrier to sync multiple processes");
        op.addOption("bench", OPT_STRING, "", "Benchmark name", 'b');
        op.addOption("hyperq", OPT_BOOL, "0", "enable hyperq");

        addBenchmarkSpecOptions(op);

        if (!op.parse(argc, argv))
        {
            op.usage();
            return (op.HelpRequested() ? 0 : 1);
        }

        // (taeklim): Open semaphore which is used for barrier 
        // for sync of kernel launch 
        bool is_barrier = op.getOptionBool("sem");
        sem_t *sem;
        if (is_barrier) {
            sem = sem_open(SEM_NAME, O_CREAT, SEM_PERMS, INITIAL_VALUE);
            if (sem == SEM_FAILED) {
                perror("sem_open(3) error");
            }
        } else {
            sem = SEM_FAILED;
        }

        bool properties = op.getOptionBool("properties");
        bool quiet = op.getOptionBool("quiet");
        string metricsfile = op.getOptionString("metricsFile");
        string outputfile = op.getOptionString("outputFile");

        int device;
        device = op.getOptionVecInt("device")[0];
        int deviceCount;
        hipGetDeviceCount(&deviceCount);
        if (device >= deviceCount) {
            cerr << "Warning: device index: " << device <<
            " out of range, defaulting to device 0.\n";
            device = 0;
        }

        // Initialization
        EnumerateDevicesAndChoose(device, properties, quiet);
        if (properties)
        {
            return 0;
        }

        // Check CUDA feature availability
        checkCudaFeatureAvailability(op);
        
        ResultDatabase resultDB;

        // (taeklim): output file
        ofstream ofile;
        if (!outputfile.empty()) {
            printf("outputfile path: %s\n", outputfile.c_str());
            ofile.open(outputfile.c_str(), ios_base::app);
        } else {
            printf("");
        }
 
        // (taeklim): Add dummy memory for memroy limitation
        uint64_t app_mem_size_mb = op.getOptionInt("dummy");
        if (app_mem_size_mb > 0) {
            hipDeviceProp_t deviceProp;
            hipGetDeviceProperties(&deviceProp, 0);
            uint64_t max_mem_size = deviceProp.totalGlobalMem / 1024 / 1024;
            printf("max_mem_size:%ld\n", max_mem_size);
            //uint64_t dummy_size_mb = MAX_MEM_SIZE - app_mem_size_mb;
            uint64_t dummy_size_mb = max_mem_size - app_mem_size_mb;

            uint64_t oversub_mb = app_mem_size_mb / op.getOptionFloat("oversub-frac");
            //dummy_size_mb = (MAX_MEM_SIZE - oversub_mb) * BYTE_PER_MB;
            dummy_size_mb = (max_mem_size - oversub_mb) * BYTE_PER_MB;

            size_t *dummy_h, *dummy;
            dummy_h = (size_t*)malloc(dummy_size_mb);
            checkCudaErrors(hipMalloc((void**)&dummy, dummy_size_mb));
            checkCudaErrors(hipMemcpy(dummy, dummy_h, dummy_size_mb, hipMemcpyHostToDevice));
            printf("Done allocating dummy memory %ld\n", dummy_size_mb);
            sleep(5);
        }

        // Run the benchmark
        RunBenchmark(resultDB, op, ofile, sem);
        printf("Done RunBenchmark\n");
        fflush(stdout);

        // If quiet, output overall result
        // else output metrics
        if (quiet) {
            resultDB.DumpOverall();
        } else {
            if (metricsfile.empty()) {
                cout << endl;
                resultDB.DumpSummary(cout);
            } else {
                ofstream ofs;
                ofs.open(metricsfile.c_str());
                resultDB.DumpCsv(metricsfile);
                ofs.close();
            }
        }
        // (taeklim)
        if (is_barrier && sem_unlink(SEM_NAME) < 0)
            perror("sem_unlink(3) failed");
    }
    catch( std::exception& e )
    {
        std::cerr << e.what() << std::endl;
        ret = 1;
    }
    catch( ... )
    {
        ret = 1;
    }

    return ret;
}
